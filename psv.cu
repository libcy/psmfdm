#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hipfft/hipfft.h>
#include "psv.h"
#include "const.h"
#include "hip/hip_runtime.h"
#include "plot.h"
#include "malloc.h"

/*
 我不慎把最终的程序弄丢了。这个程序是一个稍早的版本，有几个nx和ny搞混了，只能算nx=ny的情况
 我在老师的电脑上运行有点问题，但在我的电脑里可以正常运行，还没找到什么原因
 编译命令：nvcc psv.cu arch=sm_50 -lcufft
*/

int main(int argc , char *argv[]) {
	//output file name
	char *oname="opsv";
	char *wname="wpsv";
	int i,j;
	int calculate,out,wav;
	
	//参数中c代表计算，o代表输出波场快照，w代表输出地表地震图
	if(argc==2){
		calculate=0;
		out=0;
		wav=0;
		for(i=0;i<argv[1][i]!='\0';i++){
			if(argv[1][i]=='o') out=1;
			else if(argv[1][i]=='w') wav=1;
			else if(argv[1][i]=='c') calculate=1;
		}
	}
	else{
		calculate=1;
		out=0;
		wav=1;
	}

	//calculate
	if(calculate){
		FILE *wfile=fopen(wname,"w");
		FILE *ofile=fopen(oname,"w");

		// dimension
		//cudaMat：统一内存开辟的连续存储的二维数组，有优化空间
		float **sxx=cudaMat(nx,ny),**sxy=cudaMat(nx,ny),**syy=cudaMat(nx,ny);
		float **den=cudaMat(nx2,ny2),**rig=cudaMat(nx2,ny2),**lam=cudaMat(nx2,ny2);

		float **ux=cudaMat(nx,ny),**uy=cudaMat(nx,ny);
		float **vx=cudaMat(nx,ny),**vy=cudaMat(nx,ny);
		float **up=floatMat(nx,ny),**us=floatMat(nx,ny);

		float **dxux=cudaMat(nx,ny),**dxuy=cudaMat(nx,ny);
		float **dyux=cudaMat(nx,ny),**dyuy=cudaMat(nx,ny);
		float **dxvx=cudaMat(nx,ny),**dxvy=cudaMat(nx,ny);
		float **dyvx=cudaMat(nx,ny),**dyvy=cudaMat(nx,ny);
		float **dxsxx=cudaMat(nx,ny),**dxsxy=cudaMat(nx,ny);
		float **dysxy=cudaMat(nx,ny),**dysyy=cudaMat(nx,ny);

		float **ggg=cudaMat(nx,ny);
		float **dvp=floatMat(2,nx),**dvs=floatMat(2,nx),**dden=floatMat(2,nx);
		float **cxwork=cudaMat(nx,nx),**cywork=cudaMat(nx,nx);
		float **uxall=floatMat(nst,ntskp),**uyall=floatMat(nst,ntskp);
		float *gx=floatVec(nx),*gy=floatVec(ny);

		int *istx=intVec(nst),*isty=intVec(nst),**imap=intMat(nx,ny);
		float *nd=floatVec(nx2),*q1d=floatVec(nx2);

		for(i=0;i<nst;i++){
			istx[i]=i*4+1;
			isty[i]=na+1;
		}

		FILE *n4096=fopen("N4096.dat","r");
		FILE *q14096=fopen("Q14096.dat","r");
		for(i=0;i<nx2;i++){
			fscanf(n4096,"%f",&nd[i]);
			fscanf(q14096,"%f",&q1d[i]);
		}
		fclose(n4096);
		fclose(q14096);

		// velocity structure

		float vpb = 1.70;
		float vsb = 0.85;
		float rob = 1.8;

		float RRIGB = rob * vsb*vsb;
		float RLANB = rob * vpb*vpb - 2.0 * RRIGB;
		float RDENB = rob;

		for(i=0;i<nx2;i++){
			for(j=0;j<ny2;j++){
				float depth=j*dy/2.0;
				if(j+1<=na*2){
					rig[i][j]=0.0;
					den[i][j]=RDENB;
					lam[i][j]=0.0;
				}
				else if(j+1==na*2+1){
					rig[i][j]=RRIGB/2.0;
					den[i][j]=RDENB/2.0;
					lam[i][j]=0.0;
				}
				else{
					if(depth<=-q1d[i]/1000.0){
						vpb = 1.70;
						vsb = 0.85;
						rob = 1.8;
					}
					else if(depth<=-nd[i]/1000.0){
						vpb = 4.0;
						vsb = 2.1;
						rob = 2.4;
					}
					else if(depth<=15.0){
						vpb = 5.8;
						vsb = 3.3;
						rob = 2.7;
					}
					else if(depth<=32.0){
						vpb = 6.4;
						vsb = 3.6;
						rob = 2.85;
					}
					else{
						vpb = 6.9;
						vsb = 3.9;
						rob = 3.1;
					}

					RRIGB = rob * vsb*vsb;
					RLANB = rob * vpb*vpb - 2.0 * RRIGB;
					RDENB = rob;

					rig[i][j]=rob*vsb*vsb;
					den[i][j]=RDENB;
					lam[i][j]=RLANB;
				}
			}
		}

		for(i=0;i<nx;i++){
			for(j=0;j<ny;j++){
				imap[i][j]=0;
			}
		}

		for(i=0;i<nst;i++){
			imap[istx[i]][isty[i]]=7;
		}

		// initialize
		int kx=nbegi2(nx);
		int ky=nbegi2(ny);

		for(i=0;i<nx;i++){
			gx[i]=dx*(i+1);
		}
		for(i=0;i<ny;i++){
			gy[i]=dy*(i+1);
		}

		float ftmax=t0+at*2;

		clear(vx,nx,ny,0.0);
		clear(vy,nx,ny,0.0);
		clear(ux,nx,ny,0.0);
		clear(uy,nx,ny,0.0);
		clear(sxx,nx,ny,0.0);
		clear(sxy,nx,ny,0.0);
		clear(syy,nx,ny,0.0);

		// absorbing boundary confition
		float apara=0.015;
		float gg;
		for(i=0;i<nx;i++){
			for(j=0;j<ny;j++){
				if(i+1<nxa){
					gg=exp(-pow(apara*(nxa-i-1),2));
				}
				else if(i+1>(nx-nxa+1)){
					gg=exp(-pow(apara*(i-nx+nxa),2));
				}
				else if(j+1>(ny-nya+1)){
					gg=exp(-pow(apara*(j-ny+nya),2));
				}
				else{
					gg=1.0;
				}
				ggg[i][j]=gg;
			}
		}
		//cuFFT库创建批量FFT计划，目前是C2C，改为R2C和C2R应该会更快
		hipfftHandle plan;
		hipfftComplex *data;
		int dimension[1]={nx};
		hipfftPlanMany(&plan,1,dimension,NULL,1,1,NULL,1,1,HIPFFT_C2C,ny*2);
		hipMallocManaged((void**)&data, sizeof(hipfftComplex)*nx*ny*2);


		//time step start
		int ntw=0;
		int ntt=0;
		float t;
		clock_t start0;
		float c0=9.0/8.0;
		float c1=1.0/24.0;

		start0=clock();
		for(int it=0;it<ntmax;it++){
			if(it%((int)ntmax/10)==0) printf("%d%%\n",10*it/((int)ntmax/10));
			ntt++;
			t=dt*it;
			ntw++;

			cudaDiffxspm<<<2*nbt,ny/nbt>>>(vx,dxvx,vy,dxvy,cxwork,cywork,nx,ny,0,ky,dx);
			hipDeviceSynchronize();
			//使用统一内存需要加hipDeviceSynchronize
			//函数中的nbt是线程块数与每个线程块中线程数的比值，是根据实际测试得到的最佳值
			cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(vy,dyvy,vx,dyvx,nx,ny,dx,dy,dt,c0,c1);
			hipDeviceSynchronize();
			cudaPrep<<<nx*nbt,ny/nbt>>>(sxx,syy,sxy,lam,rig,ggg,dxvx,dxvy,dyvx,dyvy);
			hipDeviceSynchronize();
			cudaDiffxspm<<<2*nbt,ny/nbt>>>(sxy,dxsxy,sxx,dxsxx,cxwork,cywork,nx,ny,0,ky,dx);
			hipDeviceSynchronize();
			cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(sxy,dysxy,syy,dysyy,nx,ny,dx,dy,dt,c0,c1);
			hipDeviceSynchronize();
			cudaCalc<<<nx*nbt,ny/nbt>>>(vx,vy,ux,uy,dxsxx,dxsxy,dysxy,dysyy,ggg,den,t,
				ftmax,rmxx,rmxy,rmyx,rmyy,fxx,fzz,dpxx,dpzz);
			hipDeviceSynchronize();
			
			if(ntt==nskip){
				int isx,isy,it1;
				for(int ns=0;ns<nst;ns++){
					ntt=0;
					isx=istx[ns]-1;
					isy=isty[ns]-1;
					it1=(it+1)/nskip;

					uxall[ns][it1]=ux[isx][isy];
					uyall[ns][it1]=uy[isx][isy];
				}
			}

			if(ntw==nwrite){
				ntw=0;

				cudaDiffxspm<<<2*nbt,ny/nbt>>>(ux,dxux,uy,dxuy,cxwork,cywork,nx,ny,0,ky,dx);
				hipDeviceSynchronize();
				cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(uy,dyuy,ux,dyux,nx,ny,dx,dy,dt,c0,c1);
				hipDeviceSynchronize();

				for(i=0;i<nx;i++){
					for(j=0;j<ny;j++){
						up[i][j]=dxux[i][j]+dyuy[i][j];
						us[i][j]=dxuy[i][j]-dyux[i][j];
					}
				}

				fprintMat(ofile,up,nx,ny);
				fprintMat(ofile,us,nx,ny);
			}
		}

		fprintMat(wfile,uxall,nst,ntskp);
		fprintMat(wfile,uyall,nst,ntskp);

		printf("100%%\n%.2f\n",(double)(clock()-start0)/CLOCKS_PER_SEC);
	}
		
	if(out){
		snapPSV(oname);
	}

	if(wav){
		wavePSV(wname);
	}

	return 0;
}
