#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hipfft/hipfft.h>
#include "psv.h"
#include "const.h"
#include "hip/hip_runtime.h"
#include "plot.h"
#include "malloc.h"

int main(int argc , char *argv[]) {
	//output file name
	char *oname="opsv";
	char *wname="wpsv";
	int i,j;
	int calculate,out,wav;

	if(argc==2){
		calculate=0;
		out=0;
		wav=0;
		for(i=0;i<argv[1][i]!='\0';i++){
			if(argv[1][i]=='o') out=1;
			else if(argv[1][i]=='w') wav=1;
			else if(argv[1][i]=='c') calculate=1;
		}
	}
	else{
		calculate=1;
		out=0;
		wav=1;
	}

	//calculate
	if(calculate){
		FILE *wfile=fopen(wname,"w");
		FILE *ofile=fopen(oname,"w");

		// //dimension

		float **sxx=cudaMat(nx,ny),**sxy=cudaMat(nx,ny),**syy=cudaMat(nx,ny);
		float **den=cudaMat(nx2,ny2),**rig=cudaMat(nx2,ny2),**lam=cudaMat(nx2,ny2);

		float **ux=cudaMat(nx,ny),**uy=cudaMat(nx,ny);
		float **vx=cudaMat(nx,ny),**vy=cudaMat(nx,ny);
		float **up=floatMat(nx,ny),**us=floatMat(nx,ny);

		float **dxux=cudaMat(nx,ny),**dxuy=cudaMat(nx,ny);
		float **dyux=cudaMat(nx,ny),**dyuy=cudaMat(nx,ny);
		float **dxvx=cudaMat(nx,ny),**dxvy=cudaMat(nx,ny);
		float **dyvx=cudaMat(nx,ny),**dyvy=cudaMat(nx,ny);
		float **dxsxx=cudaMat(nx,ny),**dxsxy=cudaMat(nx,ny);
		float **dysxy=cudaMat(nx,ny),**dysyy=cudaMat(nx,ny);

		float **ggg=cudaMat(nx,ny);
		float **dvp=floatMat(2,nx),**dvs=floatMat(2,nx),**dden=floatMat(2,nx);
		float **cxwork=cudaMat(nx,ny),**cywork=cudaMat(nx,ny);
		float **uxall=floatMat(nst,ntskp),**uyall=floatMat(nst,ntskp);
		float *gx=floatVec(nx),*gy=floatVec(ny);

		int *istx=intVec(nst),*isty=intVec(nst),**imap=intMat(nx,ny);

		for(i=0;i<nst;i++){
			istx[i]=i*4+1;
			isty[i]=na+1;
		}

		// velocity structure

		float VPB = 6.9;
		float VSB = 3.9;
		float ROB = 3.1;

		float RRIGB = ROB * VSB*VSB;
		float RLANB = ROB * VPB*VPB - 2.0 * RRIGB;
		float RDENB = ROB;

		for(i=0;i<nx2;i++){
			for(j=0;j<ny2;j++){
				rig[i][j]=RRIGB;
				den[i][j]=RDENB;
				lam[i][j]=RLANB;
			}
		}
		for(i=0;i<nx;i++){
			for(j=0;j<ny;j++){
				imap[i][j]=0;
			}
		}

		for(i=0;i<nst;i++){
			imap[istx[i]][isty[i]]=7;
		}

		// initialize
		int kx=nbegi2(nx);
		int ky=nbegi2(ny);

		for(i=0;i<nx;i++){
			gx[i]=dx*(i+1);
		}
		for(i=0;i<ny;i++){
			gy[i]=dy*(i+1);
		}

		float ftmax=t0+at*2;

		clear(vx,nx,ny,0.0);
		clear(vy,nx,ny,0.0);
		clear(ux,nx,ny,0.0);
		clear(uy,nx,ny,0.0);
		clear(sxx,nx,ny,0.0);
		clear(sxy,nx,ny,0.0);
		clear(syy,nx,ny,0.0);

		// absorbing boundary confition
		float apara=0.015;
		float gg;
		for(i=0;i<nx;i++){
			for(j=0;j<ny;j++){
				if(i+1<nxa){
					gg=exp(-pow(apara*(nxa-i-1),2));
				}
				else if(i+1>(nx-nxa+1)){
					gg=exp(-pow(apara*(i-nx+nxa),2));
				}
				else if(j+1>(ny-nya+1)){
					gg=exp(-pow(apara*(j-ny+nya),2));
				}
				else{
					gg=1.0;
				}
				ggg[i][j]=gg;
			}
		}

		hipfftHandle plan;
		hipfftComplex *data;
		int dimension[1]={nx};
		hipfftPlanMany(&plan,1,dimension,NULL,1,1,NULL,1,1,HIPFFT_C2C,ny*2);
		hipMallocManaged((void**)&data, sizeof(hipfftComplex)*nx*ny*2);


		//time step start
		int ntw=0;
		int ntt=0;
		float t;
		clock_t start0;
		float c0=9.0/8.0;
		float c1=1.0/24.0;

		start0=clock();
		for(int it=0;it<ntmax;it++){
			if(it%((int)ntmax/10)==0) printf("%d%%\n",10*it/((int)ntmax/10));
			ntt++;
			t=dt*it;
			ntw++;

			diffxspm(vx,dxvx,vy,dxvy,plan,data,nx,ny,dx);
			cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(vy,dyvy,vx,dyvx,nx,ny,dx,dy,dt,c0,c1);
			hipDeviceSynchronize();
			cudaPrep<<<nx*nbt,ny/nbt>>>(sxx,syy,sxy,lam,rig,ggg,dxvx,dxvy,dyvx,dyvy);
			hipDeviceSynchronize();
			diffxspm(sxy,dxsxy,sxx,dxsxx,plan,data,nx,ny,dx);
			cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(sxy,dysxy,syy,dysyy,nx,ny,dx,dy,dt,c0,c1);
			hipDeviceSynchronize();
			cudaCalc<<<nx*nbt,ny/nbt>>>(vx,vy,ux,uy,dxsxx,dxsxy,dysxy,dysyy,ggg,den,t,
				ftmax,rmxx,rmxy,rmyx,rmyy,fxx,fzz,dpxx,dpzz);
			hipDeviceSynchronize();
			
			if(ntt==nskip){
				int isx,isy,it1;
				for(int ns=0;ns<nst;ns++){
					ntt=0;
					isx=istx[ns]-1;
					isy=isty[ns]-1;
					it1=(it+1)/nskip;

					uxall[ns][it1]=ux[isx][isy];
					uyall[ns][it1]=uy[isx][isy];
				}
			}

			if(ntw==nwrite){
				ntw=0;

				diffxspm(ux,dxux,uy,dxuy,plan,data,nx,ny,dx);
				cudaFinidyyx<<<2*nx*nbt,ny/nbt>>>(uy,dyuy,ux,dyux,nx,ny,dx,dy,dt,c0,c1);
				hipDeviceSynchronize();

				for(i=0;i<nx;i++){
					for(j=0;j<ny;j++){
						up[i][j]=dxux[i][j]+dyuy[i][j];
						us[i][j]=dxuy[i][j]-dyux[i][j];
					}
				}

				fprintMat(ofile,up,nx,ny);
				fprintMat(ofile,us,nx,ny);
			}
		}

		fprintMat(wfile,uxall,nst,ntskp);
		fprintMat(wfile,uyall,nst,ntskp);

		printf("100%%\n%.2f\n",(double)(clock()-start0)/CLOCKS_PER_SEC);
	}
		
	if(out){
		snapPSV(oname);
	}

	if(wav){
		wavePSV(wname);
	}

	return 0;
}
